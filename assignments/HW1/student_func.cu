#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

// A common way to represent color images is known as RGBA - the color
// is specified by how much Red, Grean and Blue is in it.
// The 'A' stands for Alpha and is used for transparency, it will be
// ignored in this homework.

// Each channel Red, Blue, Green and Alpha is represented by one byte.
// Since we are using one byte for each color there are 256 different
// possible values for each color.  This means we use 4 bytes per pixel.

// Greyscale images are represented by a single intensity value per pixel
// which is one byte in size.

// To convert an image from color to grayscale one simple method is to
// set the intensity to the average of the RGB channels.  But we will
// use a more sophisticated method that takes into account how the eye
// perceives color and weights the channels unequally.

// The eye responds most strongly to green followed by red and then blue.
// The NTSC (National Television System Committee) recommends the following
// formula for color to greyscale conversion:

// I = .299f * R + .587f * G + .114f * B

// Notice the trailing f's on the numbers which indicate that they are
// single precision floating point constants and not double precision
// constants.

// You should fill in the kernel as well as set the block and grid sizes
// so that the entire image is processed.

#include "utils.h"

__global__ void rgba_to_greyscale(const uchar4 *const rgbaImage,
                                  unsigned char *const greyImage, int numRows,
                                  int numCols) {
  // Fill in the kernel to convert from color to greyscale
  // the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  // The output (greyImage) at each pixel should be the result of
  // applying the formula: output = .299f * R + .587f * G + .114f * B;
  // Note: We will be ignoring the alpha channel for this conversion

  // First create a mapping from the 2D block and grid locations
  // to an absolute 2D location in the image, then use that to
  // calculate a 1D offset
  int offset = 0;
  int thread_grid_before = gridDim.y * blockDim.y * blockIdx.x;
  int thread_line_before = gridDim.y * blockDim.y * threadIdx.x;
  int thread_line_offset = gridDim.y * blockIdx.y + threadIdx.y;
  offset = thread_grid_before + thread_line_before + thread_line_offset;
  uchar4 rgba = rgbaImage[offset];
  float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
  greyImage[offset] = (char)channelSum;
}

void your_rgba_to_greyscale(const uchar4 *const h_rgbaImage,
                            uchar4 *const d_rgbaImage,
                            unsigned char *const d_greyImage, size_t numRows,
                            size_t numCols) {
  // Image size: 313 x 557 pixels
  // Each block has 1 thread that handles 1 pixel
  const dim3 blockSize(1, 1, 1);
  const dim3 gridSize(313, 557, 1);
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows,
                                             numCols);

  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}
